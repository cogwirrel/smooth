#include "hip/hip_runtime.h"
#ifndef CUDASMOOTH_H
#define CUDASMOOTH_H

#include "SVD2x2.cuh"

#include <hip/hip_runtime.h>
//TODO How do i use cuda standard library?
extern "C" {

__constant__ double* coords;
__constant__ double* metric;
__constant__ double* normals;
// __constant__ real_t * quality;
__constant__ size_t* ENList;
__constant__ size_t* NNListArray;
__constant__ size_t* NNListIndex;
__constant__ size_t* NEListArray;
__constant__ size_t* NEListIndex;
__constant__ int* orientation;


__device__ bool isCornerNode(size_t vid) {
  return fabs(normals[2*vid])==1.0 && fabs(normals[2*vid+1])==1.0;
}

__device__ bool isSurfaceNode(size_t vid) {
  int NE_size = NEListIndex[vid + 1] - NEListIndex[vid];
  int NN_size = NNListIndex[vid + 1] - NNListIndex[vid];
  return NE_size < NN_size;
}

__device__ double element_area(size_t eid) {
  const size_t *n = &ENList[3*eid];

  //Pointers to the coorindates of each vertex
  const double *c0 = &coords[2*n[0]];
  const double *c1 = &coords[2*n[1]];
  const double *c2 = &coords[2*n[2]];

  return *orientation * 0.5 *
          ((c0[1] - c2[1]) * (c0[0] - c1[0]) -
           (c0[1] - c1[1]) * (c0[0] - c2[0]));
}

__device__ double element_quality(size_t eid) {
  const size_t *n = &ENList[3*eid];

  // Pointers to the coordinates of each vertex
  const double *c0 = &coords[2*n[0]];
  const double *c1 = &coords[2*n[1]];
  const double *c2 = &coords[2*n[2]];

  // Pointers to the metric tensor at each vertex
  const double *m0 = &metric[3*n[0]];
  const double *m1 = &metric[3*n[1]];
  const double *m2 = &metric[3*n[2]];

  // Metric tensor averaged over the element
  double m00 = (m0[0] + m1[0] + m2[0])/3;
  double m01 = (m0[1] + m1[1] + m2[1])/3;
  double m11 = (m0[2] + m1[2] + m2[2])/3;

  // l is the length of the perimeter, measured in metric space
  double l =
    sqrt((c0[1] - c1[1])*((c0[1] - c1[1])*m11 + (c0[0] - c1[0])*m01) +
         (c0[0] - c1[0])*((c0[1] - c1[1])*m01 + (c0[0] - c1[0])*m00))+
    sqrt((c0[1] - c2[1])*((c0[1] - c2[1])*m11 + (c0[0] - c2[0])*m01) +
         (c0[0] - c2[0])*((c0[1] - c2[1])*m01 + (c0[0] - c2[0])*m00))+
    sqrt((c2[1] - c1[1])*((c2[1] - c1[1])*m11 + (c2[0] - c1[0])*m01) +
         (c2[0] - c1[0])*((c2[1] - c1[1])*m01 + (c2[0] - c1[0])*m00));

  // Area in physical space
  double a = element_area(eid);

  // Area in metric space
  double a_m = a*sqrt(m00*m11 - m01*m01);

  // Function
  double f = min(l/3.0, 3.0/l);
  double F = pow(f * (2.0 - f), 3.0);

  // This is the 2D Lipnikov functional.
  double quality = 12.0 * sqrt(3.0) * a_m * F / (l*l);

  return quality;
}

//TODO: WHat are the params?
__global__ void smooth(const size_t* colourSet, const size_t NNodesInSet) {

  const size_t threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= NNodesInSet)
    return;

  size_t vid = colourSet[threadID];

  if(isCornerNode(vid)) {
    return;
  }

  // Find the quality of the worst element adjacent to vid
  double worst_q=1.0;
  // for(std::set<size_t>::const_iterator it=mesh->NEList[vid].begin();
      // it!=mesh->NEList[vid].end(); ++it){
    // worst_q = std::min(worst_q, mesh->element_quality(*it));
  // }

  // Find begining of each vid
  for (int ne_index = NEListIndex[vid];
       ne_index < NEListIndex[vid + 1];
       ++ne_index) {
    double quality = element_quality(NEListArray[ne_index]);
    if (quality < worst_q) {
      worst_q = quality;
    }
  }

  /* Find the barycentre (centre of mass) of the cavity. A cavity is
   * defined as the set containing vid and all its adjacent vertices and
   * triangles. Since we work on metric space, all lengths have to measured
   * using the metric. The metric tensor is a 2x2 symmetric matrix which
   * encodes the ideal length and orientation of an edge containing vid. As
   * an edge is defined by two vertices, we calculate the edge length using
   * the value of the metric in the middle of the edge, i.e. the average of
   * the two metric tensors of the vertices defining the edge.
   */

  const double * m0 = &metric[3*vid];

  double x0 = coords[2*vid];
  double y0 = coords[2*vid+1];

  double A[4] = {0.0, 0.0, 0.0, 0.0};
  double q[2] = {0.0, 0.0};

  // Iterate over all edges and assemble matrices A and q.
  // for(std::vector<size_t>::const_iterator it=mesh->NNList[vid].begin();
  //     it!=mesh->NNList[vid].end(); ++it){
  for (int nn_index = NNListIndex[vid];
       nn_index < NNListIndex[vid + 1];
       ++nn_index) {

      size_t il = NNListArray[nn_index];

      const double *m1 = &metric[3*il];

      // Find the metric in the middle of the edge.
      double ml00 = 0.5*(m0[0] + m1[0]);
      double ml01 = 0.5*(m0[1] + m1[1]);
      double ml11 = 0.5*(m0[2] + m1[2]);

      double x = coords[2*il] - x0;
      double y = coords[2*il+1] - y0;

      // Calculate and accumulate the contribution of
      // this vertex to the barycentre of the cavity.
      q[0] += (ml00*x + ml01*y);
      q[1] += (ml01*x + ml11*y);

      A[0] += ml00;
      A[1] += ml01;
      A[3] += ml11;
    }

    // The metric tensor is symmetric, i.e. ml01=ml10, so A[2]=A[1].
    A[2]=A[1];

    // Displacement vector for vid
    double p[2];

  /* The displacement p for vid is found by solving the linear system:
   * ┌─       ─┐   ┌    ┐   ┌    ┐
   * │A[0] A[1]│   │p[0]│   │q[0]│
   * │         │ x │    │ = │    │
   * │A[2] A[3]│   │p[1]│   │q[0]│
   * └─       ─┘   └    ┘   └    ┘
   */
   svd_solve_2x2(A, p, q);

  /* If this is a surface vertex, restrict the displacement
   * to the surface. The new displacement is the projection
   * of the old displacement on the surface.
   */
  if(isSurfaceNode(vid)){
    p[0] -= p[0]*fabs(normals[2*vid]);
    p[1] -= p[1]*fabs(normals[2*vid+1]);
  }

  // Update the coordinates
  coords[2*vid] += p[0];
  coords[2*vid+1] += p[1];

  /************************************************************************
   * At this point we must also interpolate the metric tensors from all   *
   * neighbouring vertices in order to calculate the new value of vid's   *
   * metric tensor at the new location. This is a quite complex procedure *
   * and has been omitted for simplicity of the exercise. A vertex will   *
   * always use its original metric tensor, no matter whether it has been *
   * relocated or not.                                                    *
   ************************************************************************/

  /* Find the quality of the worst element after smoothing. If an element
   * of the cavity was inverted, i.e. if vid was relocated outside the
   * interior convex hull of the cavity, then the calculated area of that
   * element will be negative and mesh->element_quality() will return a
   * negative number. In such a case, the smoothing operation has to be
   * rejected.
   */
  double new_worst_q=1.0;
  // for(std::set<size_t>::const_iterator it=mesh->NEList[vid].begin();
      // it!=mesh->NEList[vid].end(); ++it){
    // new_worst_q = std::min(new_worst_q, mesh->element_quality(*it));
  // }
  for (int ne_index = NEListIndex[vid];
       ne_index < NEListIndex[vid + 1];
       ++ne_index) {
    double quality = element_quality(NEListArray[ne_index]);
    if (quality < new_worst_q) {
      new_worst_q = quality;
    }
  }


  /* If quality is worse than before, either because of element inversion
   * or just because relocating vid to the barycentre of the cavity does
   * not improve quality, revert the changes.
   */
  if(new_worst_q < worst_q){
    coords[2*vid] -= p[0];
    coords[2*vid+1] -= p[1];
  }
}
}

#endif
// void runCudaImplementation(Mesh* mesh, std::vector<size_t>* vids) {
//         /***********************************************/
//   // Device copy of vids
//   size_t* d_vids;

//   // Size of vids
//   size_t vid_size = vids->size() * sizeof(size_t);

//   // Allocate space for vids on device
//   hipMalloc((void **)&d_vids, vid_size);

//   // Copy host vids to device d_vids
//   hipMemcpy(d_vids, &vids[0], vid_size, hipMemcpyHostToDevice);

//   // Device copy of mesh
//   Mesh* d_mesh;
//   size_t mesh_size = sizeof(mesh);
//   hipMalloc((void**)&d_mesh, mesh_size);
//   hipMemcpy(d_mesh, mesh, mesh_size, hipMemcpyHostToDevice);

//   // Kick off parallel execution - one block per vid in vids
//   smooth_vector<<<vids->size(), 1>>>(d_mesh, d_vids);

//   // Copy result back to host
//   hipMemcpy(mesh, d_mesh, mesh_size, hipMemcpyDeviceToHost);

//   // Clean up everything bar result
//   free(vids);
//   hipFree(d_vids);
//   hipFree(d_mesh);
// }

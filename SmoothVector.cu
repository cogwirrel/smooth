#include "hip/hip_runtime.h"
#ifndef CUDASMOOTH_H
#define CUDASMOOTH_H

#include "SVD2x2.cuh"

#include <hip/hip_runtime.h>
//TODO How do i use cuda standard library?
extern "C" {

__constant__ float* coords;
__constant__ float* metric;
__constant__ float* normals;
// __constant__ real_t * quality;
__constant__ size_t* ENList;
__constant__ size_t* NNListArray;
__constant__ size_t* NNListIndex;
__constant__ size_t* NEListArray;
__constant__ size_t* NEListIndex;
__constant__ int orientation;


__device__ bool isCornerNode(size_t vid) {
  return fabs(normals[2*vid])==1.0 && fabs(normals[2*vid+1])==1.0;
}

__device__ bool isSurfaceNode(size_t vid) {
  int NE_size = NEListIndex[vid + 1] - NEListIndex[vid];
  int NN_size = NNListIndex[vid + 1] - NNListIndex[vid];
  return NE_size < NN_size;
}

__device__ float element_area(size_t eid) {
  const size_t *n = &ENList[3*eid];

  //Pointers to the coorindates of each vertex
  const float *c0 = &coords[2*n[0]];
  const float *c1 = &coords[2*n[1]];
  const float *c2 = &coords[2*n[2]];

  return orientation * 0.5 *
          ((c0[1] - c2[1]) * (c0[0] - c1[0]) -
           (c0[1] - c1[1]) * (c0[0] - c2[0]));
}

__device__ float element_quality(size_t eid) {
  const size_t *n = &ENList[3*eid];

  // Pointers to the coordinates of each vertex
  const float *c0 = &coords[2*n[0]];
  const float *c1 = &coords[2*n[1]];
  const float *c2 = &coords[2*n[2]];

  // Pointers to the metric tensor at each vertex
  const float *m0 = &metric[3*n[0]];
  const float *m1 = &metric[3*n[1]];
  const float *m2 = &metric[3*n[2]];

  // Metric tensor averaged over the element
  float m00 = (m0[0] + m1[0] + m2[0])/3;
  float m01 = (m0[1] + m1[1] + m2[1])/3;
  float m11 = (m0[2] + m1[2] + m2[2])/3;

  // l is the length of the perimeter, measured in metric space
  float l =
    sqrt((c0[1] - c1[1])*((c0[1] - c1[1])*m11 + (c0[0] - c1[0])*m01) +
         (c0[0] - c1[0])*((c0[1] - c1[1])*m01 + (c0[0] - c1[0])*m00))+
    sqrt((c0[1] - c2[1])*((c0[1] - c2[1])*m11 + (c0[0] - c2[0])*m01) +
         (c0[0] - c2[0])*((c0[1] - c2[1])*m01 + (c0[0] - c2[0])*m00))+
    sqrt((c2[1] - c1[1])*((c2[1] - c1[1])*m11 + (c2[0] - c1[0])*m01) +
         (c2[0] - c1[0])*((c2[1] - c1[1])*m01 + (c2[0] - c1[0])*m00));

  // Area in physical space
  float a = element_area(eid);

  // Area in metric space
  float a_m = a*sqrt(m00*m11 - m01*m01);

  // Function
  float f = min(l/3.0, 3.0/l);
  float F = pow(f * (2.0 - f), 3.0);

  // This is the 2D Lipnikov functional.
  float quality = 12.0 * sqrt(3.0) * a_m * F / (l*l);

  return quality;
}

//TODO: WHat are the params?
__global__ void smooth(const size_t* colourSet, const size_t NNodesInSet) {

  const size_t threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= NNodesInSet)
    return;

  size_t vid = colourSet[threadID];

  // Find the quality of the worst element adjacent to vid
  float worst_q=1.0;
  // for(std::set<size_t>::const_iterator it=mesh->NEList[vid].begin();
      // it!=mesh->NEList[vid].end(); ++it){
    // worst_q = std::min(worst_q, mesh->element_quality(*it));
  // }

  // Find begining of each vid
  for (int ne_index = NEListIndex[vid];
       ne_index < NEListIndex[vid + 1];
       ++ne_index) {
    worst_q = min(worst_q, element_quality(NEListArray[ne_index]));
  }

  // Find the barycentre (centre of mass) of the cavity. A cavity is
  // defined as the set containing vid and all its adjacent vertices and
  // triangles. Since we work on metric space, all lengths have to measured
  // using the metric. The metric tensor is a 2x2 symmetric matrix which
  // encodes the ideal length and orientation of an edge containing vid. As
  // an edge is defined by two vertices, we calculate the edge length using
  // the value of the metric in the middle of the edge, i.e. the average of
  // the two metric tensors of the vertices defining the edge.


  const float * m0 = &metric[3*vid];

  float x0 = coords[2*vid];
  float y0 = coords[2*vid+1];

  float A[4] = {0.0, 0.0, 0.0, 0.0};
  float q[2] = {0.0, 0.0};

  // Iterate over all edges and assemble matrices A and q.
  // for(std::vector<size_t>::const_iterator it=mesh->NNList[vid].begin();
  //     it!=mesh->NNList[vid].end(); ++it){
  for (int nn_index = NNListIndex[vid];
       nn_index < NNListIndex[vid + 1];
       ++nn_index) {

      size_t il = NNListArray[nn_index];

      const float *m1 = &metric[3*il];

      // Find the metric in the middle of the edge.
      float ml00 = 0.5*(m0[0] + m1[0]);
      float ml01 = 0.5*(m0[1] + m1[1]);
      float ml11 = 0.5*(m0[2] + m1[2]);

      float x = coords[2*il] - x0;
      float y = coords[2*il+1] - y0;

      // Calculate and accumulate the contribution of
      // this vertex to the barycentre of the cavity.
      q[0] += (ml00*x + ml01*y);
      q[1] += (ml01*x + ml11*y);

      A[0] += ml00;
      A[1] += ml01;
      A[3] += ml11;
    }

    // The metric tensor is symmetric, i.e. ml01=ml10, so A[2]=A[1].
    A[2]=A[1];

    // Displacement vector for vid
    float p[2];

  /// The displacement p for vid is found by solving the linear system:
  // ┌─       ─┐   ┌    ┐   ┌    ┐
  // │A[0] A[1]│   │p[0]│   │q[0]│
  // │         │ x │    │ = │    │
  // │A[2] A[3]│   │p[1]│   │q[0]│
  // └─       ─┘   └    ┘   └    ┘
  //
   svd_solve_2x2(A, p, q);

  // If this is a surface vertex, restrict the displacement
  // to the surface. The new displacement is the projection
  // of the old displacement on the surface.
  //
  if(isSurfaceNode(vid)){
    p[0] -= p[0]*fabs(normals[2*vid]);
    p[1] -= p[1]*fabs(normals[2*vid+1]);
  }

  // Update the coordinates
  coords[2*vid] += p[0];
  coords[2*vid+1] += p[1];

  /////////////////////////////////////////////////////////////////////////
  // At this point we must also interpolate the metric tensors from all   /
  // neighbouring vertices in order to calculate the new value of vid's   /
  // metric tensor at the new location. This is a quite complex procedure /
  // and has been omitted for simplicity of the exercise. A vertex will   /
  // always use its original metric tensor, no matter whether it has been /
  // relocated or not.                                                    /
  /////////////////////////////////////////////////////////////////////////

  // Find the quality of the worst element after smoothing. If an element
  // of the cavity was inverted, i.e. if vid was relocated outside the
  // interior convex hull of the cavity, then the calculated area of that
  // element will be negative and mesh->element_quality() will return a
  // negative number. In such a case, the smoothing operation has to be
  // rejected.
  //
  float new_worst_q=1.0;
  // for(std::set<size_t>::const_iterator it=mesh->NEList[vid].begin();
      // it!=mesh->NEList[vid].end(); ++it){
    // new_worst_q = std::min(new_worst_q, mesh->element_quality(*it));
  // }
  for (int ne_index = NEListIndex[vid];
       ne_index < NEListIndex[vid + 1];
       ++ne_index) {
    new_worst_q = min(new_worst_q, element_quality(NEListArray[ne_index]));
  }


  // If quality is worse than before, either because of element inversion
  // or just because relocating vid to the barycentre of the cavity does
  // not improve quality, revert the changes.
  //
  if(new_worst_q < worst_q){
    coords[2*vid] -= p[0];
    coords[2*vid+1] -= p[1];
  }
}
}

#endif


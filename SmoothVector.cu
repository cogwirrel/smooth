#include "hip/hip_runtime.h"
#include "SVD2x2.hpp"
//TODO How do i use cuda standard library?

// TODO: Do we need some special indexing here?
bool isCornerNode(size_t vid) {
  return abs(CUDA_normals[2*vid])==1.0 && abs(CUDA_normals[2*vid+1]==1.0);
}

//TODO: Do we need some special indexing here?
bool isSurfaceNode(size_t vid) {
  return CUDA_NEList[vid].size() < CUDA_NNList[vid].size();
}

double element_area(size_t eid) {
  const size_t *n = &CUDA_ENList[3*eid];

  //Pointers to the coorindates of each vertex
  const double *c0 = &CUDA_coords[2*n[0]];
  const double *c1 = &CUDA_coords[2*n[1]];
  const double *c2 = &CUDA_coords[2*n[2]];

  return orientation * 0.5 * 
          ((c0[1] - c2[1]) * (c0[0] - c1[0]) -
           (c0[1] - c1[1]) * (c0[0] - c2[0]));
}

double element_quality(size_t eid) {
  const size_t *n = &CUDA_ENList[3*eid];

  // Pointers to the coordinates of each vertex
  const double *c0 = &CUDA_coords[2*n[0]];
  const double *c1 = &CUDA_coords[2*n[1]];
  const double *c2 = &CUDA_coords[2*n[2]];

  // Pointers to the metric tensor at each vertex
  const double *m0 = &CUDA_metric[3*n[0]];
  const double *m1 = &CUDA_metric[3*n[1]];
  const double *m2 = &CUDA_metric[3*n[2]];

  // Metric tensor averaged over the element
  double m00 = (m0[0] + m1[0] + m2[0])/3;
  double m01 = (m0[1] + m1[1] + m2[1])/3;
  double m11 = (m0[2] + m1[2] + m2[2])/3;

  // l is the length of the perimeter, measured in metric space
  double l =
    sqrt((c0[1] - c1[1])*((c0[1] - c1[1])*m11 + (c0[0] - c1[0])*m01) +
         (c0[0] - c1[0])*((c0[1] - c1[1])*m01 + (c0[0] - c1[0])*m00))+
    sqrt((c0[1] - c2[1])*((c0[1] - c2[1])*m11 + (c0[0] - c2[0])*m01) +
         (c0[0] - c2[0])*((c0[1] - c2[1])*m01 + (c0[0] - c2[0])*m00))+
    sqrt((c2[1] - c1[1])*((c2[1] - c1[1])*m11 + (c2[0] - c1[0])*m01) +
         (c2[0] - c1[0])*((c2[1] - c1[1])*m01 + (c2[0] - c1[0])*m00));

  // Area in physical space
  double a = element_area(eid);

  // Area in metric space
  double a_m = a*sqrt(m00*m11 - m01*m01);

  // Function
  double f = min(l/3.0, 3.0/l);
  double F = pow(f * (2.0 - f), 3.0);

  // This is the 2D Lipnikov functional.
  double quality = 12.0 * sqrt(3.0) * a_m * F / (l*l);

  return quality;
}

//TODO: WHat are the params?
__global__ void smooth() {
  size_t vid = vids[blockIdx.x];
  if(isCornerNode(vid))
    return;

  // Find the quality of the worst element adjacent to vid
  double worst_q=1.0;
  // for(std::set<size_t>::const_iterator it=mesh->NEList[vid].begin();
      // it!=mesh->NEList[vid].end(); ++it){
    // worst_q = std::min(worst_q, mesh->element_quality(*it));
  // }

  // Find begining of each vid
  for (int ne_index = CUDA_NEIndex[ne_start]; 
       ne_index < CUDA_NEIndex[ne_start + 1]; 
       ++ne_index) {
    double quality = element_quality(CUDA_NEListArray[ne_index]);
    if (quality < worst_q) {
      worst_q = quality;
    }
  }

  /* Find the barycentre (centre of mass) of the cavity. A cavity is
   * defined as the set containing vid and all its adjacent vertices and
   * triangles. Since we work on metric space, all lengths have to measured
   * using the metric. The metric tensor is a 2x2 symmetric matrix which
   * encodes the ideal length and orientation of an edge containing vid. As
   * an edge is defined by two vertices, we calculate the edge length using
   * the value of the metric in the middle of the edge, i.e. the average of
   * the two metric tensors of the vertices defining the edge.
   */

  const double * m0 = CUDA_metric[3*vid];

  double x0 = CUDA_coords[2*vid];
  double y0 = CUDA_coords[2*vid+1];

  double A[4] = {0.0, 0.0, 0.0, 0.0};
  double q[2] = {0.0, 0.0};

  // Iterate over all edges and assemble matrices A and q.
  // for(std::vector<size_t>::const_iterator it=mesh->NNList[vid].begin();
  //     it!=mesh->NNList[vid].end(); ++it){
  for (int nn_index = CUDA_NNIndex[vid];
       nn_index < CUDA_NNIndex[vid + 1];
       ++nn_index) {
      
      size_t il = CUDA_NNListArray[nn_index];

      const double *m1 = &mesh->metric[3*il];

      // Find the metric in the middle of the edge.
      double ml00 = 0.5*(m0[0] + m1[0]);
      double ml01 = 0.5*(m0[1] + m1[1]);
      double ml11 = 0.5*(m0[2] + m1[2]);

      double x = CUDA_coords[2*il] - x0;
      double y = CUDA_coords[2*il+1] - y0;

      // Calculate and accumulate the contribution of
      // this vertex to the barycentre of the cavity.
      q[0] += (ml00*x + ml01*y);
      q[1] += (ml01*x + ml11*y);

      A[0] += ml00;
      A[1] += ml01;
      A[3] += ml11;
    }

    // The metric tensor is symmetric, i.e. ml01=ml10, so A[2]=A[1].
    A[2]=A[1];

    // Displacement vector for vid
    double p[2];

    /* The displacement p for vid is found by solving the linear system:
     * ┌─       ─┐   ┌    ┐   ┌    ┐
     * │A[0] A[1]│   │p[0]│   │q[0]│
     * │         │ x │    │ = │    │
     * │A[2] A[3]│   │p[1]│   │q[0]│
     * └─       ─┘   └    ┘   └    ┘
     */
    svd_solve_2x2(A, p, q);

    /* If this is a surface vertex, restrict the displacement
     * to the surface. The new displacement is the projection
     * of the old displacement on the surface.
     */
    if(isSurfaceNode(vid)){
      p[0] -= p[0]*fabs(CUDA_normals[2*vid]);
      p[1] -= p[1]*fabs(CUDA_normals[2*vid+1]);
    }

    // Update the coordinates
    CUDA_coords[2*vid] += p[0];
    CUDA_coords[2*vid+1] += p[1];

    /************************************************************************
     * At this point we must also interpolate the metric tensors from all   *
     * neighbouring vertices in order to calculate the new value of vid's   *
     * metric tensor at the new location. This is a quite complex procedure *
     * and has been omitted for simplicity of the exercise. A vertex will   *
     * always use its original metric tensor, no matter whether it has been *
     * relocated or not.                                                    *
     ************************************************************************/

    /* Find the quality of the worst element after smoothing. If an element
     * of the cavity was inverted, i.e. if vid was relocated outside the
     * interior convex hull of the cavity, then the calculated area of that
     * element will be negative and mesh->element_quality() will return a
     * negative number. In such a case, the smoothing operation has to be
     * rejected.
     */
    double new_worst_q=1.0;
    // for(std::set<size_t>::const_iterator it=mesh->NEList[vid].begin();
        // it!=mesh->NEList[vid].end(); ++it){
      // new_worst_q = std::min(new_worst_q, mesh->element_quality(*it));
    // }
    for (int ne_index = CUDA_NEIndex[ne_start]; 
         ne_index < CUDA_NEIndex[ne_start + 1]; 
         ++ne_index) {
      double quality = element_quality(CUDA_NEListArray[ne_index]);
      if (quality < new_worst_q) {
        new_worst_q = quality;
      }
    }


    /* If quality is worse than before, either because of element inversion
     * or just because relocating vid to the barycentre of the cavity does
     * not improve quality, revert the changes.
     */
    if(new_worst_q < worst_q){
      CUDA_coords[2*vid] -= p[0];
      CUDA_coords[2*vid+1] -= p[1];
    }
  } 
}

// void runCudaImplementation(Mesh* mesh, std::vector<size_t>* vids) {
//         /***********************************************/
//   // Device copy of vids
//   size_t* d_vids;

//   // Size of vids
//   size_t vid_size = vids->size() * sizeof(size_t);

//   // Allocate space for vids on device
//   hipMalloc((void **)&d_vids, vid_size);

//   // Copy host vids to device d_vids
//   hipMemcpy(d_vids, &vids[0], vid_size, hipMemcpyHostToDevice);
  
//   // Device copy of mesh
//   Mesh* d_mesh;
//   size_t mesh_size = sizeof(mesh);
//   hipMalloc((void**)&d_mesh, mesh_size);
//   hipMemcpy(d_mesh, mesh, mesh_size, hipMemcpyHostToDevice);

//   // Kick off parallel execution - one block per vid in vids
//   smooth_vector<<<vids->size(), 1>>>(d_mesh, d_vids);

//   // Copy result back to host
//   hipMemcpy(mesh, d_mesh, mesh_size, hipMemcpyDeviceToHost);

//   // Clean up everything bar result
//   free(vids);
//   hipFree(d_vids);
//   hipFree(d_mesh);
// }
